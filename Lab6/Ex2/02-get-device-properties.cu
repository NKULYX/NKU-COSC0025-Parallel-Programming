
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    /*
     * Assign values to these variables so that the output string below prints the
     * requested properties of the currently active GPU.
     */

    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);

    int computeCapabilityMajor = props.major;
    int computeCapabilityMinor = props.minor;
    int multiProcessorCount = props.multiProcessorCount;
    int warpSize = props.warpSize;

    /*
     * There should be no need to modify the output string below.
     */

    printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}
