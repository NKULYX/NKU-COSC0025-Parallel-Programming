#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int *a, int *b, int *c)
{
    int tid = blockIdx.x * blockDim.x * threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < N; i += stride)
        c[i] = 2 * a[i] + b[i];
}

int main()
{
    int *a, *b, *c;

    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    int size = N * sizeof(int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // Initialize memory
    for (int i = 0; i < N; ++i)
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    int threads_per_block = 512;
    int number_of_blocks = numberOfSMs * 32;

    saxpy<<<number_of_blocks, threads_per_block>>>(a, b, c);

    // Print out the first and last 5 values of c for a quality check
    for (int i = 0; i < 5; ++i)
        printf("c[%d] = %d, ", i, c[i]);
    printf("\n");
    for (int i = N - 5; i < N; ++i)
        printf("c[%d] = %d, ", i, c[i]);
    printf("\n");

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
