
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Initialize array values on the host.
 */

void init(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        a[i] = i;
    }
}

/*
 * Double elements in parallel on the GPU.
 */

__global__ void doubleElements(int *a, int N)
{
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        a[i] *= 2;
    }
}

/*
 * Check all elements have been doubled on the host.
 */

bool checkElementsAreDoubled(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        if (a[i] != i * 2)
            return false;
    }
    return true;
}

int main()
{
    int N = 100;
    int *a;

    size_t size = N * sizeof(int);

    /*
     * Refactor this memory allocation to provide a pointer
     * `a` that can be used on both the host and the device.
     */
    hipMallocManaged(&a, size);
    // a = (int *)malloc(size);

    init(a, N);

    size_t threads_per_block = 10;
    size_t number_of_blocks = 10;

    /*
     * This launch will not work until the pointer `a` is also
     * available to the device.
     */

    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    hipDeviceSynchronize();

    bool areDoubled = checkElementsAreDoubled(a, N);
    printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

    /*
     * Refactor to free memory that has been allocated to be
     * accessed by both the host and the device.
     */

    hipFree(a);
}
