
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

__global__ void firstParallel()
{
    printf("This should be running in parallel.\n");
}

int main()
{
    /*
     * Refactor this call to firstParallel to execute in parallel
     * on the GPU.
     */

    firstParallel<<<5, 5>>>();
    hipDeviceSynchronize();

    /*
     * Some code is needed below so that the CPU will wait
     * for the GPU kernels to complete before proceeding.
     */
}