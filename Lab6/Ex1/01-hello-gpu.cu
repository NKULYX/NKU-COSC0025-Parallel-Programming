
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
    printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */

__global__ void helloGPU()
{
    printf("Hello from the GPU.\n");
}

int main()
{

    helloGPU<<<1, 1>>>();
    hipDeviceSynchronize();

    helloCPU();

    /*
     * Refactor this call to `helloGPU` so that it launches
     * as a kernel on the GPU.
     */

    helloGPU<<<1, 1>>>();
    hipDeviceSynchronize();
    /*
     * Add code below to synchronize on the completion of the
     * `helloGPU` kernel completion before continuing the CPU
     * thread.
     */
}
