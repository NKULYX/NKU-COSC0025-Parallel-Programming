
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop(int N)
{
    for (int i = 0; i < N; ++i)
    {
        printf("This is iteration number %d in block %d thrad %d\n", threadIdx.x + blockIdx.x * blockDim.x, blockIdx.x, threadIdx.x);
    }
}

int main()
{
    /*
     * When refactoring `loop` to launch as a kernel, be sure
     * to use the execution configuration to control how many
     * "iterations" to perform.
     *
     * For this exercise, be sure to use more than 1 block in
     * the execution configuration.
     */

    int N = 10;
    loop<<<2, 5>>>(N);
    hipDeviceSynchronize();
}
