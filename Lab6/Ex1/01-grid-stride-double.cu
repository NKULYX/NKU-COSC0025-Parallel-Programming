
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        a[i] = i;
    }
}

/*
 * In the current application, `N` is larger than the grid.
 * Refactor this kernel to use a grid-stride loop in order that
 * each parallel thread work on more than one element of the array.
 */

__global__ void doubleElements(int *a, int N)
{
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (; i < N; i += step)
    {
        a[i] *= 2;
    }
}

bool checkElementsAreDoubled(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        if (a[i] != i * 2)
            return false;
    }
    return true;
}

int main()
{
    /*
     * `N` is greater than the size of the grid (see below).
     */

    int N = 10000;
    int *a;

    size_t size = N * sizeof(int);
    hipMallocManaged(&a, size);

    init(a, N);

    /*
     * The size of this grid is 256*32 = 8192.
     */

    size_t threads_per_block = 256;
    size_t number_of_blocks = 32;

    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    hipDeviceSynchronize();

    bool areDoubled = checkElementsAreDoubled(a, N);
    printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

    hipFree(a);
}
