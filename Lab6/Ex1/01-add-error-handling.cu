
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        a[i] = i;
    }
}

__global__ void doubleElements(int *a, int N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < N + stride; i += stride)
    {
        a[i] *= 2;
    }
}

bool checkElementsAreDoubled(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        if (a[i] != i * 2)
            return false;
    }
    return true;
}

int main()
{
    /*
     * Add error handling to this source code to learn what errors
     * exist, and then correct them. Googling error messages may be
     * of service if actions for resolving them are not clear to you.
     */

    hipError_t err;

    int N = 10000;
    int *a;

    size_t size = N * sizeof(int);
    err = hipMallocManaged(&a, size);
    if (err != hipSuccess)
    {
        printf("cuda memory error occur : %s\n", hipGetErrorString(err));
    }

    init(a, N);

    size_t threads_per_block = 2048;
    size_t number_of_blocks = 32;

    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("cuda kernel function error occur : %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("cuda synchronize error occur : %s\n", hipGetErrorString(err));
    }

    bool areDoubled = checkElementsAreDoubled(a, N);
    printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

    hipFree(a);
}
