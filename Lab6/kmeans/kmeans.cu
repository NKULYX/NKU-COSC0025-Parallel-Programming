#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"

static inline int nextPowerOfTwo(int n) {
    n--;

    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;
//  n = n >> 32 | n;    //  For 64-bit ints

    return ++n;
}

/*----< euclid_dist_2() >----------------------------------------------------*/
__host__ __device__ inline static
float euclid_dist_2(int    numCoords,
                    int    numObjs,
                    int    numClusters,
                    float *objects,
                    float *clusters,
                    int    objectId,
                    int    clusterId)
{
    int i;
    float ans=0.0;

    for (i = 0; i < numCoords; i++) {
        ans += (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]) *
               (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
    }

    return(ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *objects,
                          float *deviceClusters,
                          int *membership,
                          int *intermediates)
{
    extern __shared__ char sharedMemory[];

    unsigned char *membershipChanged = (unsigned char *)sharedMemory;
#if BLOCK_SHARED_MEM_OPTIMIZATION
    float *clusters = (float *)(sharedMemory + blockDim.x);
#else
    float *clusters = deviceClusters;
#endif

    membershipChanged[threadIdx.x] = 0;

#if BLOCK_SHARED_MEM_OPTIMIZATION
    for (int i = threadIdx.x; i < numClusters; i += blockDim.x) {
        for (int j = 0; j < numCoords; j++) {
            clusters[numClusters * j + i] = deviceClusters[numClusters * j + i];
        }
    }
    __syncthreads();
#endif

    int objectId = blockDim.x * blockIdx.x + threadIdx.x;

    if (objectId < numObjs) {
        int   index, i;
        float dist, min_dist;

        index    = 0;
        min_dist = euclid_dist_2(numCoords, numObjs, numClusters,
                                 objects, clusters, objectId, 0);

        for (i=1; i<numClusters; i++) {
            dist = euclid_dist_2(numCoords, numObjs, numClusters,
                                 objects, clusters, objectId, i);
            if (dist < min_dist) {
                min_dist = dist;
                index    = i;
            }
        }

        if (membership[objectId] != index) {
            membershipChanged[threadIdx.x] = 1;
        }

        membership[objectId] = index;

        __syncthreads();

        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                membershipChanged[threadIdx.x] +=
                    membershipChanged[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            intermediates[blockIdx.x] = membershipChanged[0];
        }
    }
}

__global__ static
void compute_delta(int *deviceIntermediates,
                   int numIntermediates,
                   int numIntermediates2)
{
    extern __shared__ unsigned int intermediates[];
    intermediates[threadIdx.x] =
        (threadIdx.x < numIntermediates) ? deviceIntermediates[threadIdx.x] : 0;

    __syncthreads();
    for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        deviceIntermediates[0] = intermediates[0];
    }
}

/*----< cuda_kmeans() >-------------------------------------------------------*/
float** cuda_kmeans(float **objects,
                   int     numCoords,
                   int     numObjs,
                   int     numClusters,
                   float   threshold,
                   int    *membership,
                   int    *loop_iterations)
{
    int      i, j, index, loop=0;
    int     *newClusterSize;
    float    delta;
    float  **dimObjects;
    float  **clusters;
    float  **dimClusters;
    float  **newClusters;

    float *deviceObjects;
    float *deviceClusters;
    int *deviceMembership;
    int *deviceIntermediates;

    malloc2D(dimObjects, numCoords, numObjs, float);
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j][i];
        }
    }

    malloc2D(dimClusters, numCoords, numClusters, float);
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }

    for (i=0; i<numObjs; i++) membership[i] = -1;

    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    malloc2D(newClusters, numCoords, numClusters, float);
    memset(newClusters[0], 0, numCoords * numClusters * sizeof(float));

    const unsigned int numThreadsPerClusterBlock = 128;
    const unsigned int numClusterBlocks =
        (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
#if BLOCK_SHARED_MEM_OPTIMIZATION
    const unsigned int clusterBlockSharedDataSize =
        numThreadsPerClusterBlock * sizeof(unsigned char) +
        numClusters * numCoords * sizeof(float);

    hipDeviceProp_t deviceProp;
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipGetDeviceProperties(&deviceProp, deviceNum);

    if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
        err("WARNING: Your CUDA hardware has insufficient block shared memory. "
            "You need to recompile with BLOCK_SHARED_MEM_OPTIMIZATION=0. "
            "See the README for details.\n");
    }
#else
    const unsigned int clusterBlockSharedDataSize =
        numThreadsPerClusterBlock * sizeof(unsigned char);
#endif

    const unsigned int numReductionThreads =
        nextPowerOfTwo(numClusterBlocks);
    const unsigned int reductionBlockSharedDataSize =
        numReductionThreads * sizeof(unsigned int);

    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&deviceIntermediates, numReductionThreads*sizeof(unsigned int)));

    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));

    do {
        checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                  numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));

        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, deviceIntermediates);

        hipDeviceSynchronize(); checkLastCudaError();

        compute_delta <<< 1, numReductionThreads, reductionBlockSharedDataSize >>>
            (deviceIntermediates, numClusterBlocks, numReductionThreads);

        hipDeviceSynchronize(); checkLastCudaError();

        int d;
        checkCuda(hipMemcpy(&d, deviceIntermediates,
                  sizeof(int), hipMemcpyDeviceToHost));
        delta = (float)d;

        checkCuda(hipMemcpy(membership, deviceMembership,
                  numObjs*sizeof(int), hipMemcpyDeviceToHost));

        for (i=0; i<numObjs; i++) {
            index = membership[i];

            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[j][index] += objects[i][j];
        }


        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
                newClusters[j][i] = 0.0;
            }
            newClusterSize[i] = 0;
        }

        delta /= numObjs;
    } while (delta > threshold && loop++ < 500);

    *loop_iterations = loop + 1;

    malloc2D(clusters, numClusters, numCoords, float);
    for (i = 0; i < numClusters; i++) {
        for (j = 0; j < numCoords; j++) {
            clusters[i][j] = dimClusters[j][i];
        }
    }

    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));
    checkCuda(hipFree(deviceIntermediates));

    free(dimObjects[0]);
    free(dimObjects);
    free(dimClusters[0]);
    free(dimClusters);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return clusters;
}
