
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printNumber(int number)
{
    printf("%d\n", number);
}

int main()
{
    for (int i = 0; i < 5; ++i)
    {
        printNumber<<<1, 1>>>(i);
    }
    hipDeviceSynchronize();
}
